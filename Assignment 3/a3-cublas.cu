#include "hip/hip_runtime.h"
#include "a3.h"
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>

void print_mat (float *A, uint16_t N) {
  for (uint16_t i = 0; i < N; i++) {
    for (uint16_t j = 0; j < N; j++) {
      printf("%.4f\t", A[i + N * j]);
    }
    printf("\n");
  }
}

int main(int argc, char **argv)
{
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipEvent_t verfStart, verfStop, start, stop;
    float total, elapsed, elapsed1;

    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(prng, 100);

    float alpha = 1.0f;
    float beta = 0.0f;

    float *dA;
    float *dB;
    float *dC;
    float *dX;
    float *dF;
    float *dError;

    hipblasHandle_t handle = 0;

    uint16_t N = 10000;
    uint16_t N2 = 2 * N;
    float sum = 0.0;

    dim3 blockDim1(1024, 1, 1); 
    dim3 gridDim1(ceil((float)(N2 * N2) / (float)blockDim1.x), 1, 1);

    //float *Q = (float *)malloc(N2 * N2 * sizeof(float));
    CHECK(hipEventCreate(&verfStart));
    CHECK(hipEventCreate(&verfStop));
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipEventRecord(verfStart, 0));
    CHECK(hipEventRecord(start, 0));
    // Create the cuBLAS handle
    CHECK_CUBLAS(hipblasCreate(&handle));

    CHECK(hipMalloc((void **)&dX, sizeof(float) * N * N));

    hiprandGenerateUniform(prng, dX, N * N);

    // Allocate device memory for vectors and the dense form for the matrices
    CHECK(hipMalloc((void **)&dA, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dA, 0, N2 * N2));
    gen_i<<<gridDim1, blockDim1>>>(dA, N2);
    CHECK_CUBLAS(hipblasSetMatrix(N, N, sizeof(float), dX, N, (dA + 2 * N * N), N2));
    CHECK(hipDeviceSynchronize());

    CHECK(hipMalloc((void **)&dB, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dB, 0, N2 * N2));
    gen_i<<<gridDim1, blockDim1>>>(dB, N2);
    alpha = -1.0f;
    CHECK_CUBLAS(hipblasSgeam(handle, trans, trans, N2, N,
                 &alpha,
                 (dB + 2 * N * N), N2,
                 &beta,
                 (dB + 2 * N * N), N2,
                 (dB + 2 * N * N), N2));
    CHECK_CUBLAS(hipblasSetMatrix(N, N, sizeof(float), dX, N, (dB + 2 * N * N), N2));
    CHECK(hipDeviceSynchronize());

    alpha = 2.0f;
    CHECK_CUBLAS(hipblasSgeam(handle, trans, trans, N, N,
                 &alpha,
                 (dB + 2 * N * N), N2,
                 &beta,
                 (dB + 2 * N * N), N2,
                 (dB + 2 * N *N), N2));
    CHECK(hipDeviceSynchronize());

    CHECK(hipFree(dX));

    CHECK(hipMalloc((void **)&dC, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dC, 0, N2 * N2));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for init1:\t %3.1f ms\n", elapsed); 

    CHECK(hipEventRecord(start, 0));
    alpha = 1.0f;
    beta = 1.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, trans, trans, N2, N2, N2,
                             &alpha,
                             dA, N2,
                             dB, N2,
                             &beta,
                             dC, N2));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for mm1:\t %3.1f ms\n", elapsed); 

    CHECK(hipEventRecord(start, 0));
    alpha = -1.0f;
    beta = 0.0f;
    CHECK_CUBLAS(hipblasSgeam(handle, trans, trans, N, N,
                 &alpha,
                 (dA + 2 * N * N), N2,
                 &beta,
                 (dA + 2 * N * N), N2,
                 (dA + 2 * N *N), N2));
    CHECK(hipDeviceSynchronize());

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for init2:\t %3.1f ms\n", elapsed); 

    CHECK(hipEventRecord(start, 0));
    CHECK(hipFree(dB));
    CHECK(hipMalloc((void **)&dB, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dB, 0.0, N2 * N2));
    alpha = 1.0f;
    beta = 1.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, trans, trans, N2, N2, N2,
                             &alpha,
                             dC, N2,
                             dA, N2,
                             &beta,
                             dB, N2));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for mm2:\t %3.1f ms\n", elapsed); 

    CHECK(hipEventRecord(start, 0));
    CHECK(hipFree(dA));
    CHECK(hipFree(dC));

    CHECK(hipMalloc((void **)&dF, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dF, 0, N2 * N2));
    gen_i<<<gridDim1, blockDim1>>>(dF, N2);
    alpha = -1.0f;
    beta = 0.0f;
    CHECK_CUBLAS(hipblasSgeam(handle, trans, trans, N2, N,
                 &alpha,
                 (dF + 2 * N * N), N2,
                 &beta,
                 (dF + 2 * N * N), N2,
                 (dF + 2 * N * N), N2));

    CHECK(hipMalloc((void **)&dError, sizeof(float) * N2 * N2));
    reduce<<<gridDim1, blockDim1, 2 * blockDim1.x * sizeof(float)>>>(dB, dF, dError, N2);
    float *err = (float *) calloc(gridDim1.x, sizeof(float));
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed1, start, stop));

    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(err, dError, sizeof(float) * gridDim1.x, hipMemcpyDeviceToHost));
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken to memcpy:\t %3.1f ms\n", elapsed);

    for (unsigned i = 0; i < gridDim1.x; i++) {
        sum += err[i];
    }
    free(err);

    printf("Time taken to verify:\t %3.1f ms\n", elapsed + elapsed1);

    CHECK(hipFree(dB));
    CHECK(hipFree(dF));
    CHECK(hipFree(dError));
    //free(Q);

    CHECK_CUBLAS(hipblasDestroy(handle));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for freeing:\t %3.1f ms\n", elapsed);

    printf("Error: %.1f\n", sum);

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(verfStop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&total, verfStart, verfStop));
    printf("Total execution time:\t %3.1f ms\n", total);

    return 0;
}
