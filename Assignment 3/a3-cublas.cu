#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>

#define THREAD_COUNT 16
#define BLOCK_DIM_X  32
#define BLOCK_DIM_Y  32

__global__ void
gen_i(float *dA, uint16_t N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        dA[i + N * i] = 1;
    }
}

void gen_identity (float *A, uint16_t N) {
  for (uint16_t i = 0; i < N; i++) {
    A[i + N * i] = 1.0;
  }
}

void gen_x (float *A, uint16_t N) {
  for (uint16_t i = 0; i < N; i++) {
    for (uint16_t j = 0; j < N; j++) {
      A[i + N * j] = rand() / (float) RAND_MAX;
    }
  }
}

void print_mat (float *A, uint16_t N) {
  for (uint16_t i = 0; i < N; i++) {
    for (uint16_t j = 0; j < N; j++) {
      printf("%.4f\t", A[i + N * j]);
    }
    printf("\n");
  }
}

void mm (float *A, float *B, float *C, uint16_t N) {
  for (uint16_t i = 0; i < N; i++)
    for (uint16_t j = 0; j < N; j++)
      for (uint16_t k = 0; k < N; k++)
        C[i + N * j] += A[i + N * k] * B[k + N * j];
}

void const_mult (float *A, int8_t c, uint16_t start_i, uint16_t start_j, uint16_t end) {
  for (uint16_t i = start_i; i < end; i++) {
    for (uint16_t j = start_j; j < end; j++) {
      A[i + end * j] = c * A[i + end * j];
    }
  }
}

void copy_submat (float *big, uint16_t bigN, float *small, uint16_t start_i, uint16_t start_j, uint16_t smallN) {
  for (uint16_t i = 0; i < smallN; i++) {
    for (uint16_t j = 0; j < smallN; j++) {
      big[(i + start_i) + bigN * (j + start_j)] = small[i + smallN * j];
    }
  }
}

int mm_eq (float *A, float *B, uint16_t N) {
  float error = 0.0;

  for (uint16_t i = 0; i < N; i++) {
    for (uint16_t j = 0; j < N; j++) {
      error += abs(A[i + N * j] - B[i + N * j]);
    }
  }

  return error;
}

int main(int argc, char **argv)
{

    dim3 blockDim1(BLOCK_DIM_X, 1);  
    hipblasOperation_t trans = HIPBLAS_OP_N;

    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(prng, 100); //(unsigned long long) clock()

    float alpha = 1.0f;
    float beta = 0.0f;

    float *dA;
    float *dB;
    float *dC;
    float *dX;

    hipblasHandle_t handle = 0;

    uint16_t N = 10000;
    uint16_t N2 = 2 * N;
    dim3 gridDim1((N2 + BLOCK_DIM_X - 1) / BLOCK_DIM_X, 1);

    // Create the cuBLAS handle
    CHECK_CUBLAS(hipblasCreate(&handle));

    CHECK(hipMalloc((void **)&dX, sizeof(float) * N * N));

    hiprandGenerateUniform(prng, dX, N * N);

    // Allocate device memory for vectors and the dense form for the matrices
    CHECK(hipMalloc((void **)&dA, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dA, 0, N2 * N2));
    gen_i<<<gridDim1, blockDim1>>>(dA, N2);
    CHECK_CUBLAS(hipblasSetMatrix(N, N, sizeof(float), dX, N, (dA + 2 * N * N), N2));
    CHECK(hipDeviceSynchronize());

    CHECK(hipMalloc((void **)&dB, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dA, 0, N2 * N2));
    gen_i<<<gridDim1, blockDim1>>>(dA, N2);
    alpha = -1.0f;
    CHECK_CUBLAS(hipblasSgeam(handle, trans, trans, N2, N,
                 &alpha,
                 (dA + 2 * N * N), N2,
                 &beta,
                 (dA + 2 * N * N), N2,
                 (dA + 2 * N * N), N2));
    CHECK_CUBLAS(hipblasSetMatrix(N, N, sizeof(float), dX, N, (dA + 2 * N * N), N2));
    alpha = 2.0f;
    CHECK_CUBLAS(hipblasSgeam(handle, trans, trans, N, N,
                 &alpha,
                 (dB + 2 * N * N), N2,
                 &beta,
                 (dB + 2 * N * N), N2,
                 (dB + 2 * N *N), N2));
    CHECK(hipDeviceSynchronize());

    CHECK(hipFree(dX));

    CHECK(hipMalloc((void **)&dC, sizeof(float) * N2 * N2));
    //CHECK(hipMalloc((void **)&dF, sizeof(float) * N2 * N2));

    CHECK_CUBLAS(hipblasSgemm(handle, trans, trans, N2, N2, N2,
                             &alpha,
                             dA, N2,
                             dB, N2,
                             &beta,
                             dC, N2));

    alpha = -1.0f;
    CHECK_CUBLAS(hipblasSgeam(handle, trans, trans, N, N,
                 &alpha,
                 (dA + 2 * N * N), N2,
                 &beta,
                 (dA + 2 * N * N), N2,
                 (dA + 2 * N *N), N2));


    CHECK_CUBLAS(hipblasSgemm(handle, trans, trans, N2, N2, N2,
                             &alpha,
                             dC, N2,
                             dA, N2,
                             &beta,
                             dB, N2));

    // Copy the result vector back to the host
//    CHECK(hipMemcpy(E, dE, sizeof(float) * N2 * N2, hipMemcpyDeviceToHost));

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));

    //printf("Error: %d\n", mm_eq(E, F, N2));

    //free(E);
    //free(F);

    //CHECK(hipFree(dF));

    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}
