#include "hip/hip_runtime.h"
#include "a3.h"
#include <hipsparse.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>

int main(int argc, char **argv)
{
    hipEvent_t verfStart, verfStop, start, stop;
    float total, elapsed, elapsed1;
    hipStream_t stream_0, stream_1;

    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(prng, 100);

    hipsparseOperation_t trans = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    hipblasOperation_t trans_blas = HIPBLAS_OP_N;
    hipblasHandle_t handle_blas = 0;
    CHECK_CUBLAS(hipblasCreate(&handle_blas));

    int *dNnzPerRowA,
      *dNnzPerRowB,
      *dNnzPerRowC,
      *dNnzPerRowE;

    float *dCsrValA,
      *dCsrValB,
      *dCsrValC,
      *dCsrValD,
      *dCsrValE;

    int *dCsrRowPtrA,
      *dCsrRowPtrB,
      *dCsrRowPtrC,
      *dCsrRowPtrD,
      *dCsrRowPtrE;

    int *dCsrColIndA,
      *dCsrColIndB,
      *dCsrColIndC,
      *dCsrColIndD,
      *dCsrColIndE;

    int totalNnzA,
      totalNnzB,
      totalNnzC,
      baseD, totalNnzD,
      baseE, totalNnzE;

    float alpha = 1.0f;
    float beta = 1.0f;

    float *dQ, *dX;
    float *dF;
    float *dError;

    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    int *nnzTotalDevHostPtr = &totalNnzD;

    uint16_t N = 10000;
    uint16_t N2 = 2 * N;
    float sum = 0.0;

    dim3 blockDim1(1024, 1, 1); 
    dim3 gridDim1(ceil((float)(N2 * N2) / (float)blockDim1.x), 1, 1);

    CHECK(hipEventCreate(&verfStart));
    CHECK(hipEventCreate(&verfStop));
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipStreamCreate(&stream_0));
    CHECK(hipStreamCreate(&stream_1));

    CHECK(hipEventRecord(verfStart, 0));
    CHECK(hipEventRecord(start, 0));

    CHECK(hipMalloc((void **)&dX, sizeof(float) * N * N));

    hiprandGenerateUniform(prng, dX, N * N);

    // [I X]
    // [O I]
    CHECK(hipMalloc((void **)&dQ, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dQ, 0, N2 * N2));
    gen_i<<<gridDim1, blockDim1>>>(dQ, N2);

    CHECK_CUBLAS(hipblasSetMatrix(N, N, sizeof(float), dX, N, (dQ + 2 * N * N), N2));

    CHECK(hipFree(dX));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for init:\t %3.1f ms\n", elapsed); 

    CHECK(hipEventRecord(start, 0));

    // Create the cuSPARSE handle
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    CHECK_CUSPARSE(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    // Construct a descriptor for the matrices
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    CHECK(hipMalloc((void **)&dNnzPerRowA, sizeof(int) * N2));
    CHECK(hipMalloc((void **)&dNnzPerRowC, sizeof(int) * N2));
    CHECK(hipMalloc((void **)&dNnzPerRowE, sizeof(int) * N2));
    CHECK(hipMalloc((void **)&dNnzPerRowB, sizeof(int) * N2));

    CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, N2, N2, descr, dQ,
                                N2, dNnzPerRowA, &totalNnzA));

    CHECK(hipMalloc((void **)&dCsrValA, sizeof(float) * totalNnzA));
    CHECK(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (N2 + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalNnzA));
    CHECK_CUSPARSE(hipsparseSdense2csr(handle, N2, N2, descr, dQ, N2, dNnzPerRowA,
                                      dCsrValA, dCsrRowPtrA, dCsrColIndA));

    // XXX: dQ: A -> C
    // [I -X]
    // [O  I]

    alpha = -1.0f;
    beta = 0.0f;
    CHECK_CUBLAS(hipblasSgeam(handle_blas, trans_blas, trans_blas, N, N,
                 &alpha,
                 (dQ + 2 * N * N), N2,
                 &beta,
                 (dQ + 2 * N * N), N2,
                 (dQ + 2 * N *N), N2));

    CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, N2, N2, descr, dQ,
                                N2, dNnzPerRowC, &totalNnzC));
    CHECK(hipMalloc((void **)&dCsrValC, sizeof(float) * totalNnzC));
    CHECK(hipMalloc((void **)&dCsrRowPtrC, sizeof(int) * (N2 + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndC, sizeof(int) * totalNnzC));

    CHECK_CUSPARSE(hipsparseSdense2csr(handle, N2, N2, descr, dQ, N2, dNnzPerRowC,
                                      dCsrValC, dCsrRowPtrC, dCsrColIndC));


    // XXX: dQ: A -> B
    // [I  X]
    // [O -I]

    alpha = -1.0f;
    CHECK_CUBLAS(hipblasSgeam(handle_blas, trans_blas, trans_blas, N2, N,
                 &alpha,
                 (dQ + 2 * N * N), N2,
                 &beta,
                 (dQ + 2 * N * N), N2,
                 (dQ + 2 * N * N), N2));

    // [I 2X]
    // [O -I]
    alpha = 2.0f;
    CHECK_CUBLAS(hipblasSgeam(handle_blas, trans_blas, trans_blas, N, N,
                 &alpha,
                 (dQ + 2 * N * N), N2,
                 &beta,
                 (dQ + 2 * N * N), N2,
                 (dQ + 2 * N *N), N2));

    CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, N2, N2, descr, dQ,
                                N2, dNnzPerRowB, &totalNnzB));

    CHECK(hipMalloc((void **)&dCsrValB, sizeof(float) * totalNnzB));
    CHECK(hipMalloc((void **)&dCsrRowPtrB, sizeof(int) * (N2 + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndB, sizeof(int) * totalNnzB));
    CHECK_CUSPARSE(hipsparseSdense2csr(handle, N2, N2, descr, dQ, N2, dNnzPerRowB,
                                      dCsrValB, dCsrRowPtrB, dCsrColIndB));

    CHECK(hipMalloc((void **)&dCsrRowPtrD, sizeof(int) * (N2 + 1)));

    CHECK_CUSPARSE(hipsparseXcsrgemmNnz(handle, trans, trans, N2, N2, N2, 
                        descr, totalNnzA, dCsrRowPtrA, dCsrColIndA,
                        descr, totalNnzB, dCsrRowPtrB, dCsrColIndB,
                        descr, dCsrRowPtrD, nnzTotalDevHostPtr));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));

    CHECK(hipEventSynchronize(stop));

    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for init2:\t %3.1f ms\n", elapsed); 

    CHECK(hipEventRecord(start, 0));
    if (NULL != nnzTotalDevHostPtr) {
        totalNnzD = *nnzTotalDevHostPtr;
    } else {
        hipMemcpy(&totalNnzD, dCsrRowPtrD+N2, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseD, dCsrRowPtrD, sizeof(int), hipMemcpyDeviceToHost);
        totalNnzD -= baseD;
    }
    hipMalloc((void**)&dCsrColIndD, sizeof(int)*totalNnzD);
    hipMalloc((void**)&dCsrValD, sizeof(float)*totalNnzD);

    // Perform matrix-vector multiplication with the CSR-formatted matrices
    CHECK_CUSPARSE(hipsparseScsrgemm(handle, trans, trans, N2, N2, N2,
                                    descr, totalNnzA, dCsrValA, dCsrRowPtrA, dCsrColIndA,
                                    descr, totalNnzB, dCsrValB, dCsrRowPtrB, dCsrColIndB,
                                    descr, dCsrValD, dCsrRowPtrD, dCsrColIndD));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for mm1:\t %3.1f ms\n", elapsed); 

    CHECK(hipEventRecord(start, 0));
    CHECK(hipFree(dCsrValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCsrColIndA));
    CHECK(hipFree(dNnzPerRowA));
    CHECK(hipFree(dCsrValB));
    CHECK(hipFree(dCsrRowPtrB));
    CHECK(hipFree(dCsrColIndB));
    CHECK(hipFree(dNnzPerRowB));

    CHECK(hipMalloc((void **)&dCsrRowPtrE, sizeof(int) * (N2 + 1)));

    CHECK_CUSPARSE(hipsparseXcsrgemmNnz(handle, trans, trans, N2, N2, N2, 
                        descr, totalNnzD, dCsrRowPtrD, dCsrColIndD,
                        descr, totalNnzC, dCsrRowPtrC, dCsrColIndC,
                        descr, dCsrRowPtrE, &totalNnzE));

    if (NULL != nnzTotalDevHostPtr) {
        totalNnzE = *nnzTotalDevHostPtr;
    } else {
        hipMemcpy(&totalNnzE, dCsrRowPtrE+N2, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseE, dCsrRowPtrE, sizeof(int), hipMemcpyDeviceToHost);
        totalNnzE -= baseE;
    }
    hipMalloc((void**)&dCsrColIndE, sizeof(int)*totalNnzE);
    hipMalloc((void**)&dCsrValE, sizeof(float)*totalNnzE);

    CHECK_CUSPARSE(hipsparseScsrgemm(handle, trans, trans, N2, N2, N2,
                                    descr, totalNnzD, dCsrValD, dCsrRowPtrD, dCsrColIndD,
                                    descr, totalNnzC, dCsrValC, dCsrRowPtrC, dCsrColIndC,
                                    descr, dCsrValE, dCsrRowPtrE, dCsrColIndE));

    CHECK_CUSPARSE(hipsparseScsr2dense(handle, N2, N2, descr,
                                      dCsrValE, dCsrRowPtrE, dCsrColIndE,
                                      dQ, N2));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for mm2:\t %3.1f ms\n", elapsed); 

    // XXX: CHECK RESULTS
    
    CHECK(hipEventRecord(start, 0));

    CHECK(hipFree(dCsrValC));
    CHECK(hipFree(dCsrRowPtrC));
    CHECK(hipFree(dCsrColIndC));
    CHECK(hipFree(dNnzPerRowC));
    CHECK(hipFree(dCsrValD));
    CHECK(hipFree(dCsrRowPtrD));
    CHECK(hipFree(dCsrColIndD));
    CHECK(hipFree(dCsrValE));
    CHECK(hipFree(dCsrRowPtrE));
    CHECK(hipFree(dCsrColIndE));
    CHECK(hipFree(dNnzPerRowE));
    
    CHECK(hipMalloc((void **)&dF, sizeof(float) * N2 * N2));
    CHECK(hipMemset(dF, 0, N2 * N2));
    gen_i<<<gridDim1, blockDim1>>>(dF, N2);
    alpha = -1.0f;
    CHECK_CUBLAS(hipblasSgeam(handle_blas, trans_blas, trans_blas, N2, N,
                 &alpha,
                 (dF + 2 * N * N), N2,
                 &beta,
                 (dF + 2 * N * N), N2,
                 (dF + 2 * N * N), N2));

    CHECK(hipMalloc((void **)&dError, sizeof(float) * N2 * N2));
    reduce<<<gridDim1, blockDim1, 2 * blockDim1.x * sizeof(float)>>>(dQ, dF, dError, N2);
    float *err = (float *) calloc(gridDim1.x, sizeof(float));
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed1, start, stop));

    CHECK(hipEventRecord(start, 0));
    CHECK(hipMemcpy(err, dError, sizeof(float) * gridDim1.x, hipMemcpyDeviceToHost));
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken to memcpy:\t %3.1f ms\n", elapsed);

    for (unsigned i = 0; i < gridDim1.x; i++) {
        sum += err[i];
    }
    free(err);

    CHECK(hipFree(dQ));
    CHECK(hipFree(dF));
    CHECK(hipFree(dError));

    printf("Time taken to verify:\t %3.1f ms\n", elapsed + elapsed1);

    CHECK(hipEventRecord(start, 0));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken for freeing:\t %3.1f ms\n", elapsed); 

    printf("Error: %.1f\n", sum);

    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(verfStop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&total, verfStart, verfStop));
    printf("Total execution time:\t %3.1f ms\n", total);

    return 0;
}
